#include "hip/hip_runtime.h"
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128	
#define INLINE inline __device__ __host__
#include <cassert>
#include "common.h"
#include <vector>
#include <algorithm>
#include <iterator>
#include <fstream>
#include <limits>
#include <iostream>
#include <hip/hip_runtime.h>
//#if __CUDA_ARCH__ >= 200
//#include <sm_20_atomic_functions.h>
//#if __CUDA_ARCH__ > 100      // Atomics only used with > sm_10 architecture
#include "cuPrintf.cu"
#include <sm_20_atomic_functions.h>
//#endif

//#define DEBUG_OUT

#define MAX_BLOCK_SIZE 256
#define CHECK_POINT //printf("CHECK POINT:\t%s\t%d\n",__FILE__,__LINE__);

__global__ void EchoKernel(){
	cuPrintf("Echo from GPU\n");
}

inline __host__ __device__ int3 make_int3(int4 a)
{
    return make_int3(a.x, a.y, a.z);
}

SGNTR_CUDA_ECHO{
	printf("ECHO[::\n");
	cudaPrintfInit();
	dim3 dimGrid(2, 2);
	dim3 dimBlock(2, 2, 2);
	EchoKernel<<<dimGrid, dimBlock>>>();
	hipDeviceSynchronize();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	hipDeviceReset();
	printf("::]ENDECHO\n");
}

inline __host__ __device__ int4 make_int4(int x,int y,int z)
{
    return make_int4(x, y, z, 0);
}

namespace cuda_side{
	typedef uchar3 pixel_type;
	pixel_type *devImg = 0;//column major
	pixel_type *hostImg = 0;//column major
	pixel_type *img = 0;//column major
	unsigned img_sz_in_byt = 0;//in bytes
	unsigned rows = 0;// the 'inner' dimension
	unsigned cols = 0;// the 'outer' dimension
	unsigned img_pitch_in_px = 0;// >=rows
	__device__ __constant__ float2 half_viewport_size;
	__device__ __constant__ float viewport_aspect;
	
	__device__ float4 preprocess_sum_accum;
	__device__ float4 preprocess_normsq_accum;
	float4 norm_u_pr_t_k; // ||u'(t,*,*,k)||

#define ZERO_PARTIAL_SUM make_int4(0,0,0,0)
#define MAKE_PARTIAL_SUM make_int4
	typedef int4 partial_sum_type;
	partial_sum_type *line_partial_sum = 0;//column major
	texture<partial_sum_type,2,hipReadModeElementType> l_p_s_tex_ref;
	//texture<partial_sum_type, 2, hipReadModeElementType> lps_tex_ref;
	//size_t l_p_s_sz = 0;
	size_t l_p_s_pitch = 0;//in sizeof(partial_sum_type) instead of bytes


	unsigned num_verts = 0;
	vertex_type *world_space_vertices = 0;
	vertex_type *transformed_vertices = 0;

#define MAX_BATCH_SIZE 16
	__device__ __constant__ transform_type wvp_transforms[MAX_BATCH_SIZE];
	__device__ float opt_obj_gpu_acc[MAX_BATCH_SIZE*16/*13*/];
	float opt_obj_gpu_acc_memsetter[MAX_BATCH_SIZE*16/*13*/];

	typedef float4 scanline_accum_type;
#define ZERO_SCANLINE_ACCUM make_float4(0,0,0,0);

/*	//colors
	typedef double3 color_type;
	const unsigned color_count = 5;
	color_type colors[color_count];

	//color output
#define ZERO_COLOR_ACCUM make_int4(0,0,0,0)
#define MAKE_COLOR_ACCUM make_int4
	typedef int4 color_accum_type;//must be signed
	color_accum_type *color_accum = 0;//the innermost dimension can be the batch size(MAX_BATCH_SIZE>1) or the triangle count(MAX_BATCH_SIZE==1)
	unsigned color_accum_bases[color_count+1];
	typedef uint3 c_a_lim_type;
	c_a_lim_type *c_a_lims = 0;
#if(MAX_BATCH_SIZE>1)
	size_t c_a_pitch = 0;//in sizeof(color_accum_type) instead of bytes
#endif */

	unsigned tri_count = 0;
	vertex_type *tri_verts = 0;
	typedef float3 color_type;
	color_type *tri_colors = 0;

};
using namespace cuda_side;

void Ensure(hipError_t rc,int line){
	if(rc!=hipSuccess){
		fprintf(stderr,"%d:%s!=hipSuccess at line %d.\n",rc,hipGetErrorString(rc),line);
		
		assert(0);
	}
}
#define SANITY0 \
				{float buf[1024]; \
				ENSURE(hipMemcpy(buf,tri_verts,tri_count*3*sizeof(vertex_type),hipMemcpyDeviceToHost)); \
				for(int i=0;i<tri_count*3;++i){ \
					printf("%f ",buf[i]); \
				} printf("\n");}

SGNTR_CUDA_SETUP{
	assert(copy_to_device||page_locked_mem);
	assert(!cuda_array||copy_to_device);
	{
		float2 s = {0.5f * (cols = width),0.5f * (rows = height)};
		img_pitch_in_px = ((rows+63)/64)*64;
		printf("%d %d\n",img_pitch_in_px,rows);
		assert(img_pitch_in_px>=rows); 
		assert(img_pitch_in_px<rows+64);
		img_sz_in_byt = img_pitch_in_px * cols * sizeof(pixel_type);
		ENSURE(hipMemcpyToSymbol(HIP_SYMBOL(half_viewport_size),&s,sizeof(s)));
		float a = double(width)/double(height);
		ENSURE(hipMemcpyToSymbol(HIP_SYMBOL(viewport_aspect),&a,sizeof(a)));
	}
	if(copy_to_device){
		if(cuda_array){
			assert(0);
		}else{
			ENSURE(hipMalloc(&devImg,img_sz_in_byt));
			assert(devImg);
			img = devImg;
		}
	}
	if(page_locked_mem){
		ENSURE(hipHostAlloc( &hostImg, img_sz_in_byt, hipHostMallocWriteCombined | (copy_to_device?0:hipHostMallocMapped) ));
		assert(hostImg);
		if(!copy_to_device){
			ENSURE(hipHostGetDevicePointer(&img,hostImg,0));
		}
	}
	//l_p_s_sz = (rows+1) * (cols+1) * sizeof(partial_sum_type); // Make it a little larger than needed for the ease of modification
	//ENSURE(hipMalloc(&line_partial_sum,l_p_s_sz));
	ENSURE(hipMallocPitch(&line_partial_sum,&l_p_s_pitch,rows*sizeof(partial_sum_type),cols+1));
	assert(l_p_s_pitch>=rows*sizeof(partial_sum_type));
	assert(l_p_s_pitch%sizeof(partial_sum_type)==0);
	l_p_s_pitch/=sizeof(partial_sum_type);
	assert(line_partial_sum);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<partial_sum_type>();
	ENSURE(hipBindTexture2D(0, &l_p_s_tex_ref, line_partial_sum, &channelDesc, rows/**sizeof(partial_sum_type)*/, cols+1, l_p_s_pitch*sizeof(partial_sum_type)));
	assert(!indexed);
	using namespace std;
	{
		//(ofstream("junk.txt"))<<123;
#define _LOAD(V,T)		vector<T> V;		{	ifstream s( #V ".txt" ); copy(istream_iterator<T>(s),istream_iterator<T>(),back_inserter(V));	}
		_LOAD(verts,vertex_type);
		_LOAD(tris,tri_type);
		_LOAD(colors_,color_type); 		//there can be triangles with negative colors.And the triangle list file should already be sorted by color.
#undef _LOAD
		main_verts = verts;
		main_tris = tris;

//		assert(colors_.size()==color_count);
//		std::copy(colors_.begin(),colors_.end(),colors);

		tri_count = tris.size();

//#if(MAX_BATCH_SIZE>1)
//		ENSURE(hipMallocPitch(&color_accum,&c_a_pitch,MAX_BATCH_SIZE*sizeof(color_accum_type),color_count/*tri_count*/));
//		assert(c_a_pitch);
//		assert(c_a_pitch%sizeof(color_accum_type) == 0);
//		c_a_pitch/=sizeof(color_accum_type);
//#else
//		ENSURE(hipMalloc(&color_accum,color_count/*tri_count*/*sizeof(color_accum_type)));
//#endif
		ENSURE(hipMalloc(&tri_verts,tri_count*3*sizeof(vertex_type)));
		assert(tri_verts);
		CHECK_POINT;
		ENSURE(hipMalloc(&tri_colors,tri_count*sizeof(color_type)));
		assert(tri_colors);
//		ENSURE(hipMalloc(&c_a_lims,tri_count*sizeof(c_a_lim_type)));
//		assert(c_a_lims);
		{
			std::vector<vertex_type> tvb(tri_count*3);
			std::vector<color_type> coloring(tri_count);
//			std::vector<c_a_lim_type> calb(tri_count);
//			unsigned currentColor = 0xFFFFFFFF;
			for(unsigned i=0;i<tri_count;++i){
				assert(tris[i].w<colors_.size());
#define PUT_V(N,C) /*cout<<(i*3+N)<<' '<<i<<' '<<tris[i].C<<endl;*/ tvb[i*3+N] = verts[tris[i].C];
				PUT_V(0,x);
				PUT_V(1,y);
				PUT_V(2,z);
#undef PUT_V
				coloring[i] = colors_[tris[i].w];
			}
			for(int i=0;i<tri_count*3;++i){
				printf("%f %f %f\n",tvb[i].x,tvb[i].y,tvb[i].z);
			}
			ENSURE(hipMemcpy(tri_verts,&tvb.front(),tri_count*3*sizeof(vertex_type),hipMemcpyHostToDevice));
			ENSURE(hipMemcpy(tri_colors,&coloring.front(),tri_count*sizeof(color_type),hipMemcpyHostToDevice));
			
			//SANITY0;
			CHECK_POINT;
//			ENSURE(hipMemcpy(c_a_lims,&calb.front(),tri_count*sizeof(c_a_lim_type),hipMemcpyHostToDevice));
		}
	}
}

SGNTR_CUDA_TEARDOWN{
	if(hostImg) ENSURE(hipHostFree(hostImg));
	if(devImg) ENSURE(hipFree(devImg));
	assert(line_partial_sum); ENSURE(hipFree(line_partial_sum));
	//assert(color_accum); ENSURE(hipFree(color_accum));
	assert(tri_verts); ENSURE(hipFree(tri_verts));
	assert(tri_colors); ENSURE(hipFree(tri_colors));
	//assert(c_a_lims); ENSURE(hipFree(c_a_lims));
}

//use the following #def's to disable verbose output
#define cuPrintf(...)
#define cudaPrintfInit(...)
#define cudaPrintfDisplay(...)
#define cudaPrintfEnd(...)

#define MAP3(OP) OP(x);OP(y);OP(z);
#define MAP4(OP) OP(x);OP(y);OP(z);OP(w);
__global__ void Preprocess(
	partial_sum_type* out, //s(t)
	int out_pitch, 
	const pixel_type *in, 
	int in_row_count, 
	int pitch,
	int in_mat_size, 
	float avg_mult //(HW)^(-1)
//	double4 *normsq_out, //||u'(t,*,*,k)||^2 for k=1..4
//	int cap
	) // Preprocessing of Input Frames: evalutate s(t) and ||u'(t,*,*,k)|| for all k.
{
    int r = blockDim.x * blockIdx.x + threadIdx.x;
	if (r < in_row_count){
		const pixel_type *in_begin = in + r;
		const pixel_type *in_end = in_begin + in_mat_size;
		partial_sum_type *p_out = out + r;
		*p_out = ZERO_PARTIAL_SUM;
		for(const pixel_type *p_in=in_begin; p_in<in_end; p_in+=pitch){
			partial_sum_type *p=p_out;
			p_out+=out_pitch;
			*p_out = MAKE_PARTIAL_SUM(p_in->x,p_in->y,p_in->z,1)+*p;
			/*	cuPrintf("PP:%d\t:\t%d\t%d\t%d\t:\t%d\t%d\t%d\t%d\n",
				p_in->x, p_in->y, p_in->z, p->x, p->y, p->z, p->w);
			*/
			//partial_sum_type delta = *p;
			//cuPrintf("PP: %d\t%d\t%d\t%d",delta.x,delta.y,delta.z,delta.w);
		}
		__shared__ float buf[MAX_BLOCK_SIZE*4];
		{
			float *p=buf+threadIdx.x;
#define WT(C) *p=p_out->C;p+=blockDim.x;
			MAP4(WT);
#undef WT
			__syncthreads();
			cuPrintf("EEE:%d %d %d %d\n",blockDim.x,blockDim.y,threadIdx.x,threadIdx.y);
			cuPrintf("PP:%d %d %d %d %d: %f %f %f %f\n", r, p_out->x, p_out->y, p_out->z, p_out->w, p[-blockDim.x*4], p[-blockDim.x*3], p[-blockDim.x*2], p[-blockDim.x*1] );
		}
		//blockDim.x must be power of two
		for(int stride=blockDim.x>>1;stride>0;stride>>=1) {	
			{
				__syncthreads();
			//	float *p=buf+threadIdx.x;
			//	cuPrintf("KKK0:%d %d %d :: %f %f %f %f\n", r, threadIdx.x, stride, p[0], p[blockDim.x], p[2*blockDim.x], p[3*blockDim.x]);
			}
			if(threadIdx.x<stride && r+stride<in_row_count && threadIdx.x+stride<blockDim.x){
				float *p=buf+threadIdx.x;
				cuPrintf("KKK:%d %d %d :: %f %f %f %f\n", r, threadIdx.x, stride, p[stride], p[stride+blockDim.x], p[stride+2*blockDim.x], p[stride+3*blockDim.x]);
#define AC(C) *p+=*(p+stride); p+=blockDim.x;
				MAP4(AC);
#undef AC
			}
		}
		if(!threadIdx.x){
			float *p=buf;
			cuPrintf("QQQ:%f %f %f %f\n",p[0],p[blockDim.x],p[blockDim.x*2],p[blockDim.x*3]);
#define A(C) atomicAdd(&preprocess_sum_accum.C,*p);	p+=blockDim.x;
			MAP4(A);
#undef A
		}
		__syncthreads();
		cuPrintf("LLL:%f %f %f %f :: %f %f %f %f\n", 
			preprocess_sum_accum.x, preprocess_sum_accum.y, preprocess_sum_accum.z, preprocess_sum_accum.w,
			preprocess_normsq_accum.x, preprocess_normsq_accum.y, preprocess_normsq_accum.z, preprocess_normsq_accum.w);
		float4 sum=preprocess_sum_accum;
		float4 avg=sum*avg_mult;
		float4 sq_sum=make_float4(0,0,0,0);
		for(const pixel_type *p_in=in_begin; p_in<in_end; p_in+=in_row_count){
#define S(C) {float t = p_in->C-avg.C; sq_sum.C+=t*t;}	
			MAP3(S);
#undef S
		}
		{
			{
				float *p=buf+threadIdx.x;
#define WT(C) *p=sq_sum.C; p+=blockDim.x;
				MAP4(WT);
#undef WT
			}
			for(int stride=blockDim.x>>1;stride>0;stride>>=1)		{
				__syncthreads();
				if(threadIdx.x<stride && r+stride<in_row_count && threadIdx.x+stride<blockDim.x){
					float *p=buf+threadIdx.x;
#define AC(C) *p+=*(p+stride); p+=blockDim.x;
					MAP4(AC);
#undef AC
				}
			}
			if(!threadIdx.x){
				float *p=buf;
#define A(C) atomicAdd(&preprocess_normsq_accum.C,*p);	p+=blockDim.x;
				MAP4(A);
#undef A
			}
		}
/*		{ __syncthreads();
		cuPrintf("MMM:%f %f %f %f :: %f %f %f %f\n", 
			preprocess_sum_accum.x, preprocess_sum_accum.y, preprocess_sum_accum.z, preprocess_sum_accum.w,
			preprocess_normsq_accum.x, preprocess_normsq_accum.y, preprocess_normsq_accum.z, preprocess_normsq_accum.w); } */
	}
}


#define RUN_KERNEL_1D(K,N,P) { \
	int threadsPerBlock = 256; assert(threadsPerBlock<=MAX_BLOCK_SIZE);\
	int blocksPerGrid = ((N) + threadsPerBlock - 1) / threadsPerBlock; \
	K<<<blocksPerGrid, threadsPerBlock>>>P; \
}

SGNTR_CUDA_UPDATE{ //Preprocessing of Input Frames
	//SANITY0;
	CHECK_POINT;
#define FIRST_COPY(DEST,DSTTYPE) \
		ENSURE(hipMemcpy2D(	DEST,	img_pitch_in_px*sizeof(pixel_type),	\
								pixels,		rows*sizeof(pixel_type),	\
								rows*sizeof(pixel_type),	cols,	cudaMemcpyHostTo##DSTTYPE));
	if(hostImg){
		FIRST_COPY(hostImg,Host);
		if(devImg){
			ENSURE(hipMemcpyAsync(devImg,hostImg,img_sz_in_byt,hipMemcpyHostToDevice));
		}
	}else{
		assert(devImg);
		FIRST_COPY(devImg,Device);
	}
	ENSURE(hipDeviceSynchronize());
	ENSURE(hipGetLastError());
	//SANITY0;
	CHECK_POINT;
	//RUN_KERNEL_OVER_ROWS(CalcPartialSum,rows,(img, line_partial_sum, rows, rows*cols));
	{
		float4 s=make_float4(0,0,0,0);
		ENSURE(hipMemcpyToSymbol(HIP_SYMBOL(preprocess_sum_accum),&s,sizeof(s)));
		ENSURE(hipMemcpyToSymbol(HIP_SYMBOL(preprocess_normsq_accum),&s,sizeof(s)));
	}
	//blockDim.x must be powers of two
	//SANITY0;
	CHECK_POINT;
	cudaPrintfInit();
	RUN_KERNEL_1D(Preprocess,rows,(line_partial_sum, l_p_s_pitch, img, rows, img_pitch_in_px, img_pitch_in_px*cols, 1.0/(rows*cols)));
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();	
	//SANITY0;
	CHECK_POINT;
	{
		float4 s;			
		ENSURE(hipMemcpyFromSymbol( &s, HIP_SYMBOL(preprocess_normsq_accum), sizeof(preprocess_normsq_accum)));
#define K(C) norm_u_pr_t_k.C = sqrt(s.C);
		MAP4(K);
#undef K
//#define P(V4) printf(#V4" = (%f,\t%f,\t%f\t,%f)\n",V4.x,V4.y,V4.z,V4.w);
//		P(norm_u_pr_t_k);
//#undef P
	}
	ENSURE(hipDeviceSynchronize());
	ENSURE(hipGetLastError());
	//SANITY0;
	CHECK_POINT;
}



__global__ void EvalObjFuncGPUPhase( // Evaluation of Optimization Objective: GPU Phase    (per triangle)
	vertex_type *tri_world_verts, 
	color_type *tri_colors,
	int rdc_stride,
//	partial_sum_type* s_t, //s(t)
	int nTri)
{//out: Z, c*Z, n_q, n_q*c, n_q*c^2
	float4 ss[4];
	int iTri = blockDim.y*blockIdx.x+threadIdx.y;
	int iConf = threadIdx.x;
	int nConf = blockDim.x;
	if(iTri>=nTri) return;
#if(MAX_BATCH_SIZE>1)
	#define TO_SS(I)	ss[I] = wvp_transforms[iConf]*make_float4(tri_world_verts[iTri*3+I],1);  \
	cuPrintf("VID:\t%d\tWORLDPOS: %f\t%f\t%f\tSCRPOS:\t%f\t%f\t%f\n",iTri*3+I,\
		tri_world_verts[iTri*3+I].x,tri_world_verts[iTri*3+I].y,tri_world_verts[iTri*3+I].z,ss[I].x,ss[I].y,ss[I].z);
#else 
	#define TO_SS(I)	ss[I] = wvp_transforms[0]*make_float4(tri_world_verts[iTri*3+I],1);
#endif

	TO_SS(0);
	TO_SS(1);
	TO_SS(2);
#undef TO_SS
	
#define CLIP(I) ((ss[I].z>0)<<I)
	unsigned clip_mask = CLIP(0)|CLIP(1)|CLIP(2);
	const uint4 clip_func_table[8] = {//x is the number of clipped points. y is "the different one".
		{0,0,1,2},//000
		{1,0,1,2},//001
		{1,1,2,0},//010
		{2,2,0,1},//011
		{1,2,0,1},//100
		{2,1,2,0},//101
		{2,0,1,2},//110
		{3,0,1,2} //111
	};//should try moving it into constant memory, probably already done by nvcc.
	uint4 clip_stat = clip_func_table[clip_mask];
#undef CLIP

//	c_a_lim_type cal = cals[iTri];
	scanline_accum_type accumulated = ZERO_SCANLINE_ACCUM;
//#define CUT(SRC,DEST) { a = ss[clip_stat.y].z/(ss[clip_stat.y].z-ss[clip_stat.SRC].z); ss[clip_stat.DEST] = ss[clip_stat.y]*(1-a)+ss[clip_stat.SRC]*a; }
#define CUT(VISIBLE,INVIS,RESULT) { float a = ss[clip_stat.VISIBLE].z/(ss[clip_stat.VISIBLE].z-ss[clip_stat.INVIS].z); \
				ss[clip_stat.RESULT] = ss[clip_stat.VISIBLE]+(ss[clip_stat.INVIS]-ss[clip_stat.VISIBLE])*a; } //should try both.
	if(3==clip_stat.x){
	}else{
		if(2==clip_stat.x){
			CUT(y,z,z);
			CUT(y,w,w);
		}
		if(1==clip_stat.x){
			clip_stat.x = 3;
			CUT(w,y,x);
			CUT(z,y,y);
		}else{
			clip_stat.x = clip_stat.w;
		}
		//float2 start,end;//already clipped?
#define PRSP(V,C) V.C/V.w
#define ACCUM(P0,P1)	\
					{	\
						float	x0 = PRSP(ss[P0],x), \
								y0 = -PRSP(ss[P0],y), \
								x1 = PRSP(ss[P1],x), \
								y1 = -PRSP(ss[P1],y); \
						float dy = y1-y0; float dxdy = (x1-x0)*viewport_aspect/dy; \
						float c; \
						float r0,r1; \
						if(y1>y0){ r0 = y0; r1 = y1; c = x0; }else{ r0 = y1; r1 = y0; c = x1; } \
						float r; \
						c = (c+1.0f)*half_viewport_size.x+modff( (r0+1.0f)*half_viewport_size.y, &r )*dxdy; \
						r1 = truncf((r1+1.0f)*half_viewport_size.y)-0.5f; \
						scanline_accum_type a = ZERO_SCANLINE_ACCUM; \
						for(; r<r1 ; r+=1.0f, c+=dxdy ){ \
							a += /*MAKE_COLOR_ACCUM*/make_float4(tex2D(l_p_s_tex_ref,r,c)); /*{int4 p = tex2D(l_p_s_tex_ref,r,c); cuPrintf("\tBBB:%f\t%f\t%d\t%d\t%d\t%d\n",r,c,p.x,p.y,p.z,p.w);}*/ \
						} /*should be auto-clamped by texture sampler*/ \
						accumulated+=a*copysignf(1.0f,dy); \
						/*cuPrintf("AAA:dy=%f\tx0=%f\ty0=%f\tx1=%f\ty1=%f\tr=%f\tr1=%f\tc=%f\n",dy,x0,y0,x1,y1,(r),(r1),(c));*/	\
						/*{ scanline_accum_type b = a*copysignf(1.0f,dy); cuPrintf("a\t%f\t%f\t%f\t%f\tb\t%f\t%f\t%f\t%f\tdy\t%f\n",a.x,a.y,a.z,a.w,b.x,b.y,b.z,b.w,dy);}*/ \
					}
		ACCUM(clip_stat.x,clip_stat.y);
		ACCUM(clip_stat.y,clip_stat.z);
		ACCUM(clip_stat.z,clip_stat.w);
		if(1==clip_stat.x){ ACCUM(clip_stat.w,clip_stat.x); }
#undef ACCUM
#undef CUT
	}

#if(MAX_BATCH_SIZE>1) 
	const unsigned tid = threadIdx.x+threadIdx.y*blockDim.x;
	#define F(x) (MAX_BATCH_SIZE*x)
#else 
	Fail to compile.
	//unsigned tid = threadIdx.y;
	//#define F(x) (x)
#endif

	cuPrintf("Echo from GPU:tid=%d,\tiTri=%d,\tiConf=%d,\taccumulated=(%f,%f,%f,%f)\n",tid,iTri,iConf, accumulated.x, accumulated.y, accumulated.z, accumulated.w);
	//OUT = accumulated;
	//tid = threadIdx.x;
	__shared__ float buf[MAX_BLOCK_SIZE*(4+3+3+3)]; //Every int4 occupies 4 banks.A separate buffer for every channel may be better.		
	int smp = blockDim.x*blockDim.y;
	color_type color = tri_colors[iTri];
	// typedef float acc_t;//can be float,double,or long long
	

	{
		// Z:3,  n_q:1, n_q*c:3, Z*c:3*2, n_q*c^2:3*2     (in 32-bit words)
		float *p = buf+tid;
#define Z accumulated
#define W0(V) *p=V;p+=smp;
#define W(C) W0(Z.C) 
		MAP4(W);//Z (as (x,y,z)) and n_q (as w)
#undef W
#define W(C) W0(Z.w*color.C) 
		MAP3(W);//n_q*c
#undef W
//#undef W0

		//acc_t *p8 = (acc_t*)p;
//#define W0(V) *p8=V;p8+=smp;
#define W(C) W0(color.C* /*(acc_t)*/ Z.C)
		MAP3(W);
#undef W
#define W(C) W0(color.C*color.C* /*(acc_t)*/ Z.w)
		MAP3(W);
#undef W
#undef W0
#undef Z
		__syncthreads();
	}
	

	/*{	
		float *p = buf+tid;
		//cuPrintf("PER_TRI_0:%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n",p[0],p[smp],p[smp*2],p[smp*3],p[smp*4],p[smp*5],p[smp*6],p[smp*7],p[smp*8],p[smp*9]);
		cuPrintf("PER_TRI_1:color=(%f,%f,%f),Z=(%f,%f,%f,%f),%f,%f,%f\n",color.x,color.y,color.z,accumulated.x,accumulated.y,accumulated.z,accumulated.w,p[smp*7],p[smp*8],p[smp*9]);
		cuPrintf("ASSERTION0:%d (%f %f %f)\n",p[smp*4]==p[smp*5] && p[smp*5]==p[smp*6],p[smp*4],p[smp*5],p[smp*6]);
#define Z accumulated
//		cuPrintf("PER_TRI_Z:(%f : %f, \t\t%f : %f, \t\t%f : %f, \t\t%f : %f)\n",p[0],Z.x, p[smp],Z.y, p[smp*2],Z.z, p[smp*3],Z.w);
		
//		cuPrintf("PER_TRI_Z:(%f : %f, \t\t%f : %f, \t\t%f : %f, \t\t%f : %f)\n",p[0],Z.x, p[smp],Z.y, p[smp*2],Z.z, p[smp*3],Z.w);
#undef Z
	}*/
	{
		//	iTri = blockDim.y*blockIdx.x+threadIdx.y;
		//iBatch = threadIdx.x;
		//tid = threadIdx.x+threadIdx.y*blockDim.x;

		int stride_check=min(nTri-iTri,blockDim.y-threadIdx.y)*nConf;		//stride<stride_check is equivalent to iTri+stride/nConf<nTri && ...<threadIdx.y
		//First stride is the smallest y such that y==(2^x)*nConf and 2*y>=nConf*blockDim.y
		//Last stride is nConf
		cuPrintf("stride_check=%d\tnTri=%d\tiTri=%d\tnConf=%d\n",stride_check,nTri,iTri,nConf);
		for(int stride=rdc_stride; stride>0 ; stride>>=1)    {
			__syncthreads();
			if(stride<stride_check && tid<stride && nConf<=stride){ 
#define A(P) {*P+=*(P+stride);P+=smp;}
				float *p = buf+tid;
				cuPrintf("RDC:STRD=%d\tTID=%d\tps=%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n",
					stride,tid,p[0],p[smp],p[2*smp],p[3*smp],p[4*smp],p[smp*5],p[smp*6],p[smp*7]);
#define D A(p)
				D;D;D;  D;  D;D;D;//Z, n_q, n_q*c
//#undef D
//			acc_t *p8 = (acc_t*)p;
//#define D A(p8)
				D;D;D;  D;D;D;//Z*c, n_q*c^2
#undef D
#undef A
		/*		{//TEST
					p=buf+tid;
					cuPrintf("ASSERTION2:%d %d %d %d %d\n",iTri+stride/nConf<nTri,iTri,stride,nConf,nTri);
					cuPrintf("ASSERTION1:%d %d (%f %f %f)@%d (%f %f %f)@%d\n",p[smp*4]==p[smp*5] && p[smp*5]==p[smp*6],stride,p[smp*4],p[smp*5],p[smp*6],p+smp*4,p[stride+smp*4],p[stride+smp*5],p[stride+smp*6],p+smp*4+stride);
				}*/
			}
		}
		__syncthreads();
		/*{//TEST
#define X (opt_obj_gpu_acc+iConf*16)
			__syncthreads();
			cuPrintf("opt_obj_gpu_acc1\t%f\t%f\t%f, \t%f, \t%f\t%f\t%f, \t%f\t%f\t%f\n",X[0],X[1],X[2],X[3],X[4],X[5],X[6],X[7],X[8],X[9]);
			__syncthreads();
#undef X
		}*/
		if(!threadIdx.y){
			float *p = buf+tid;
			float *wt = opt_obj_gpu_acc+iConf*16; // 13 "D"s follow
			cuPrintf("CCCC:%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n",p[0],p[smp],p[2*smp],p[3*smp],p[4*smp],p[5*smp],p[6*smp],p[7*smp],p[8*smp],p[9*smp]);
#define A(P)  atomicAdd(wt,/*blockIdx.x?(-0.3f):0.5f*/*P);++wt;P+=smp;
#define D A(p)
			D;D;D;	D;	D;D;D; //sum Z, sum n_q, sum n_q*c
//#undef D
//			acc_t *p8 = (acc_t*)p;
//#define D A(p8)
			D;D;D;	D;D;D; //sum Z*c, sum n_q*c^2
#undef D
#undef A
		}
		/*{//TEST
			__syncthreads();
			float *p = buf+tid;
			float *wt = opt_obj_gpu_acc+iConf*16; 
			//cuPrintf("opt_obj_gpu_acc2\t%f\t%f\t%f, \t%f, \t%f\t%f\t%f, \t%f\t%f\t%f\n",wt[0],wt[1],wt[2],wt[3],wt[4],wt[5],wt[6],wt[7],wt[8],wt[9]);
			cuPrintf("VVVVV:%d %d %d: %f\t%f\t%f\t:\t %f \t\t %f \t\t %f \n",wt+4,wt+5,wt+6, p[4*smp],p[5*smp],p[6*smp], wt[4], wt[5], wt[6]);
		}*/
	}
#undef F
}

SGNTR_CUDA_TRY_CONFIGURATION{ //Evaluation of Optimization Objective
	CHECK_POINT;
	ENSURE(hipGetLastError());
//	static std::vector<color_accum_type> cas;
//	size_t s = c_a_pitch*sizeof(color_accum_type)*color_count;
//	cas.resize(c_a_pitch*color_count);
	for(unsigned batchBase=0;batchBase<wvps.size();batchBase+=MAX_BATCH_SIZE){
		//setup matrices first
		unsigned n = std::min((size_t)MAX_BATCH_SIZE,wvps.size()-batchBase);
		unsigned m = 256/n;
		//std::copy(wvps.begin()+batchBase,wvps.begin()+batchBase+n,wvp_transforms);
		ENSURE(hipMemcpyToSymbol(HIP_SYMBOL(wvp_transforms),&wvps.front()+batchBase,sizeof(transform_type)*n));
	/*	for(int j=0;j<n;++j){
			for(int r=0;r<4;++r){
				const float4 &s = wvps[j].rows[r];
				printf("%f %f %f %f\n",s.x,s.y,s.z,s.w);
			}
			printf("--------\n");
		}*/

		assert(sizeof(opt_obj_gpu_acc)==sizeof(opt_obj_gpu_acc_memsetter));
		memset(opt_obj_gpu_acc_memsetter,0,sizeof(opt_obj_gpu_acc_memsetter));
//		ENSURE(hipMemset((void*)opt_obj_gpu_acc,100,sizeof(opt_obj_gpu_acc)));
		ENSURE(hipMemcpyToSymbol(HIP_SYMBOL(opt_obj_gpu_acc), opt_obj_gpu_acc_memsetter, sizeof(opt_obj_gpu_acc_memsetter)));
		ENSURE(hipDeviceSynchronize());
/*		{
			static float gpu_acc[MAX_BATCH_SIZE*16];
			ENSURE(hipMemcpyFromSymbol( gpu_acc, HIP_SYMBOL(opt_obj_gpu_acc), n*16, 0, hipMemcpyDeviceToHost ));
			for(int j=0;j<n;++j){
				float *f=gpu_acc+16*j;
				printf("fs: ");
				for(int k=0;k<13;++k){
					printf("%f ",f[k]);
				} 
				printf("\n"); 
			}
		}*/
	
		ENSURE(hipGetLastError());
		cudaPrintfInit();
		//ENSURE(hipMemset2D(color_accum,c_a_pitch*sizeof(color_accum_type),0,MAX_BATCH_SIZE*sizeof(color_accum_type),n));
		int rdc_stride=n;
		while(2*rdc_stride<n*m) rdc_stride<<=1;
//		printf("rdc_stride=%d\tn=%d\tm=%d\n",rdc_stride,n,m);

		{
			//SANITY0;CHECK_POINT;
			dim3 threadsPerBlock(n,m);
			assert(n*m<=MAX_BLOCK_SIZE);
			int blocksPerGrid = ((tri_count) + m - 1) / m;
			EvalObjFuncGPUPhase<<<blocksPerGrid,threadsPerBlock>>>( tri_verts, tri_colors, rdc_stride, tri_count );
			//std::cout<<threadsPerBlock.x<<' '<<threadsPerBlock.y<<' '<<threadsPerBlock.z<<' '<<blocksPerGrid<<' ';
		}
		ENSURE(hipGetLastError());
		ENSURE(hipDeviceSynchronize());
		static float gpu_acc[MAX_BATCH_SIZE*16];
		ENSURE(hipMemcpyFromSymbol( gpu_acc, HIP_SYMBOL(opt_obj_gpu_acc), n*16*sizeof(float) ));
		for(int j=0;j<n;++j){
			float *f=gpu_acc+16*j;
			float3 sum_Z=make_float3(f[0],f[1],f[2]);
			float sum_n_q(f[3]);
			float3 sum_n_q_by_c=make_float3(f[4],f[5],f[6]);
			float3 sum_Z_by_c=make_float3(f[7],f[8],f[9]);
			float3 sum_n_q_by_c_2=make_float3(f[10],f[11],f[12]);
			float3 G = sum_n_q_by_c/sum_n_q;
			float3 B = sum_n_q_by_c_2 - 2*G*sum_n_q_by_c + G*G*sum_n_q;
			float3 A = sum_Z_by_c - G*sum_Z;
			float3 tar;
#define F(K) tar.K = A.K/(sqrt(B.K)*norm_u_pr_t_k.K);
			MAP3(F);
#undef F
			//let I(k)=1
			//printf("fs: ");
			//for(int k=0;k<13;++k){
			//	printf("%f ",f[k]);
			//} 
			//printf("\n");
			//printf("sum_n_q = %f\n", sum_n_q);
#define P(V3) printf(#V3" = (%f,\t%f,\t%f)\n",V3.x,V3.y,V3.z);
			//P(sum_Z);P(sum_n_q_by_c);P(sum_Z_by_c);P(sum_n_q_by_c_2);P(G);P(B);P(A);P(tar);
#undef P
			out[batchBase+j] = tar.x+tar.y+tar.z;
			//printf("out[batchBase+j]: %f\n",out[batchBase+j]);
		}
		cudaPrintfDisplay(stdout, true);
		cudaPrintfEnd();
	//std::copy(cas.begin(),cas.begin()+c_a_pitch*color_count,std::ostream_iterator<color_accum_type>(std::cout,"\n"));
/*		for(unsigned j=0;j<n;++j){
			double3 sStencil={0,0,0};
		//std::cout<<"sStencil="<<sStencil<<"\n";
			double area = 0;
			for(unsigned k=0;k<color_count;++k){
				double a = cas[k*c_a_pitch+j].w;
				sStencil+=colors[k]*a;
				area+=a;
			//std::cout<<"	sStencil="<<sStencil<<"a="<<a<<"colors["<<k<<"]="<<colors[k]<<"colors[k]*a="<<colors[k]*a<<"\n";
			}
		//std::cout<<"sStencil="<<sStencil<<"area="<<area<<"\n";
			sStencil *= 1.0/area;
		//std::cout<<"sStencil="<<sStencil<<"\n";
			double s = 0;
			for(unsigned k=0;k<color_count;++k){
				color_accum_type ca=cas[k*c_a_pitch+j];
			//std::cout<<"colors["<<k<<"]-Stencil="<<colors[k]-sStencil<<"ca="<<ca<<"\n";
			//std::cout<<dot(colors[k]-sStencil,make_double3(ca.x,ca.y,ca.z))<<"\n";
				s+=dot(colors[k]-sStencil,make_double3(ca.x,ca.y,ca.z));
			//std::cout<<"s="<<s<<"\n";
			}
			out[i+j] = s/area;
		}*/
	}
	//std::copy(out.begin(),out.end(),std::ostream_iterator<double>(std::cout,","));std::cout<<'\n';
	//while(true);
	CHECK_POINT;
}

template<typename T>
__device__ inline void PrintI3OnDevice(const T &v){
	cuPrintf("( %d,\t%d,\t%d )",v.x,v.y,v.z);
}

template<typename T>
__device__ inline void PrintOnDevice(const T &v){
	PrintI3OnDevice(v);
}


template<typename S, typename T>
void PrintStdVectorAsRowMajMatrix(S &s, const std::vector<T> &v, unsigned r, unsigned c){
	assert(v.size() == r*c);
	for(typename std::vector<T>::const_iterator i = v.begin(); i!=v.end(); i+=c){
		copy(i,i+c,std::ostream_iterator<T>(s,"\t"));
		s<<'\n';
	}
}

template<typename T>
std::vector<T> DownloadToStdVector(const T *src_on_device, typename std::vector<T>::size_type n){
	std::vector<T> r(n);
	assert(r.size()==n);
	ENSURE(hipMemcpy( &r.front(), src_on_device, n*sizeof(T), hipMemcpyDeviceToHost ));
	return r;
}

SGNTR_CUDA_DUMP{
	//Printed matrices are transposed;
	PrintStdVectorAsRowMajMatrix(s, DownloadToStdVector(img, img_pitch_in_px*cols), cols, img_pitch_in_px);
	//PrintStdVectorAsRowMajMatrix(s, DownloadToStdVector(line_partial_sum, rows*(cols+1)), cols+1, rows);
	PrintStdVectorAsRowMajMatrix(s, DownloadToStdVector(line_partial_sum, l_p_s_pitch*(cols+1)), cols+1, l_p_s_pitch);
}
